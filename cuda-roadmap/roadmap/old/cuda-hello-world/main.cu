
#include <iostream>
#include <hip/hip_runtime.h>

// CUDA Kernel function to execute on the GPU
__global__ void helloFromGPU() {
    printf("Hello, World from GPU!\n");
}

int main() {
    std::cout << "Hello, World from CPU!\n";

    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "helloFromGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    return 0;
}

