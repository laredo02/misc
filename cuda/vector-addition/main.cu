

#include <iostream>
#include <hip/hip_runtime.h>

// CUDA Kernel for Vector Addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements) {
		C[i] = A[i] + B[i];
	}
}

int main(void) {
    int numElements = 50000; // Number of elements in the vectors
    size_t size = numElements * sizeof(float); // Size of the data

    // Allocate host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate device vectors
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy the vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Addition CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy the result from device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}



